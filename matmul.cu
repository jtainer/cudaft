#include "hip/hip_runtime.h"
// 
// Cuda kernel for transforming a vector by a matrix of complex numbers
//
// 2022, Jonathan Tainer
//

#include "matmul.h"
#include "complex.h"

__global__
void matmul(Complex* matrix, float* input, Complex* output, unsigned int N) {
	
	// Determine the thread ID
	int y = threadIdx.x + (blockIdx.x * blockDim.x);

	// Ensure excess threads do nothing
	if (y < N) {
		
		Complex sum = { 0, 0 };

		for (unsigned int x = 0; x < N; x++) {
			sum.real += matrix[(y * N) + x].real * input[x];
			sum.imag += matrix[(y * N) + x].imag * input[x];
		}

		output[y] = sum;
	}
}

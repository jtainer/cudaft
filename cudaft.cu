#include "hip/hip_runtime.h"
// 
// DFT class implementation
// 
// 2022, Jonathan Tainer
// 

#include "cudaft.h"
#include "complex.h"

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

CudaFT::CudaFT() {
	N = 0;
	devMatrix = NULL;
	devInput = NULL;
	devOutput = NULL;
}

CudaFT::~CudaFT() {
	hipFree(devMatrix);
	hipFree(devInput);
	hipFree(devOutput);
}

void CudaFT::setDimensions(unsigned int n) {
	
	// Allocate or reallocate an appropriate amount of GPU memory
	hipFree(devMatrix);
	hipFree(devInput);
	hipFree(devOutput);

	N = n;

	hipMalloc((void**)&devMatrix, sizeof(Complex) * N * N);
	hipMalloc((void**)&devInput, sizeof(float) * N);
	hipMalloc((void**)&devOutput, sizeof(Complex) * N);

	// Compute the DFT matrix in system memory
	Complex* sysMatrix = (Complex*)malloc(sizeof(Complex) * N * N);
	
	Complex omega;
	omega.real = cosf(-2 * M_PI / N);
	omega.imag = sinf(-2 * M_PI / N);

	for (unsigned int y = 0; y < N; y++) {
		for (unsigned int x = 0; x < N; x++) {
			sysMatrix[(y * N) + x] = omega.exp(x * y);
		}
	}

	// Copy DFT matrix to GPU memory
	hipMemcpy(devMatrix, sysMatrix, sizeof(Complex) * N * N, hipMemcpyHostToDevice);

	free(sysMatrix);
}

void CudaFT::phasorTransform(float* inputBuffer, Complex* outputBuffer) {
	
}
